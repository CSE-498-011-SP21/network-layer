//
// Created by depaulsmiller on 3/17/21.
//

#include <networklayer/cuda/connection.cuh>
#include <networklayer/cuda/gpu_buf.cuh>
#include <unistd.h>

int LOG_LEVEL = TRACE;

int main(int argc, char **argv) {

    cse498::unique_buf buf;

    char *gpu_buf, *cpu_buf;
    cpu_buf = new char[4096];

    hipMalloc(&gpu_buf, 4096);

    {
        DO_LOG(DEBUG) << (void *) gpu_buf;
    }

    cse498::gpu_buf remoteAccess(gpu_buf, cpu_buf, 4096);

    const char *addr = "127.0.0.1";

    if (argc > 1) {
        addr = argv[1];
    }

    auto *c1 = new cse498::Connection(addr, true, 8080, cse498::Verbs);

    while (!c1->connect());

    *((uint64_t *) remoteAccess.getCPU()) = ~0;
    remoteAccess.moveToGPU();
    uint64_t key = 1;
    c1->register_mr(remoteAccess, FI_REMOTE_WRITE | FI_REMOTE_READ, key, true);
    uint64_t key2 = 2;
    c1->register_mr(buf, FI_SEND | FI_RECV | FI_WRITE | FI_REMOTE_WRITE | FI_READ | FI_REMOTE_READ, key2);

    std::cerr << "Send\n";

    *((uint64_t *) buf.get()) = key;

    c1->send(buf, sizeof(uint64_t));

    std::cerr << "Send\n";

    *((uint64_t *) buf.get()) = (uint64_t) remoteAccess.get();

    c1->send(buf, sizeof(uint64_t));

    std::cerr << "Recv\n";

    c1->recv(buf, 1);

    return 0;
}
